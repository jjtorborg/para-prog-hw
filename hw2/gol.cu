// Para Prog: Assignment 2

// References: https://devblogs.nvidia.com/even-easier-introduction-cuda/
//      Line 285 - uses the equation presented in this article ^^^
//               -> int numBlocks = (N + blockSize - 1) / blockSize;

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>

// CUDA Includes for parallel programming
#include <hip/hip_runtime.h>


// Result from last compute of world.
unsigned char *g_resultData = NULL;

// Current state of world.
unsigned char *g_data = NULL;

// Current width of world.
size_t g_worldWidth = 0;

/// Current height of world.
size_t g_worldHeight = 0;

/// Current data length (product of width and height)
size_t g_dataLength = 0; // g_worldWidth * g_worldHeight

// Method for properly zeroing out all shared memory upon init
void sharedMemoryInit(unsigned char **d_data, size_t d_dataLength)
{
    hipMallocManaged(d_data, (d_dataLength * sizeof(unsigned char)));
    memset(*d_data, 0, sizeof(*d_data));
}

static inline void gol_initAllZeros(size_t worldWidth, size_t worldHeight)
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // init the shared memory to all zeros
    sharedMemoryInit(&g_data, g_dataLength);
    sharedMemoryInit(&g_resultData, g_dataLength);
}

static inline void gol_initAllOnes(size_t worldWidth, size_t worldHeight)
{
    int i;

    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Init the shared memory of the original grid to all zeros
    sharedMemoryInit(&g_data, g_dataLength);

    // set all rows of world to true
    for (i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 1;
    }

    // Init the shared memory of the reuslt grid to all zeros
    sharedMemoryInit(&g_resultData, g_dataLength);
}

static inline void gol_initOnesInMiddle(size_t worldWidth, size_t worldHeight)
{
    int i;

    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Init the shared memory of the original grid to all zeros
    sharedMemoryInit(&g_data, g_dataLength);

    // set first 1 rows of world to true
    for (i = 10 * g_worldWidth; i < 11 * g_worldWidth; i++)
    {
        if ((i >= (10 * g_worldWidth + 10)) && (i < (10 * g_worldWidth + 20)))
        {
            g_data[i] = 1;
        }
    }

    // Init the shared memory of the reuslt grid to all zeros
    sharedMemoryInit(&g_resultData, g_dataLength);
}

static inline void gol_initOnesAtCorners(size_t worldWidth, size_t worldHeight)
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Init the shared memory of the original grid to all zeros
    sharedMemoryInit(&g_data, g_dataLength);

    g_data[0] = 1;                                                 // upper left
    g_data[worldWidth - 1] = 1;                                    // upper right
    g_data[(worldHeight * (worldWidth - 1))] = 1;                  // lower left
    g_data[(worldHeight * (worldWidth - 1)) + worldWidth - 1] = 1; // lower right

    // Init the shared memory of the reuslt grid to all zeros
    sharedMemoryInit(&g_resultData, g_dataLength);
}

static inline void gol_initSpinnerAtCorner(size_t worldWidth, size_t worldHeight)
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Init the shared memory of the original grid to all zeros
    sharedMemoryInit(&g_data, g_dataLength);

    g_data[0] = 1;              // upper left
    g_data[1] = 1;              // upper left +1
    g_data[worldWidth - 1] = 1; // upper right

    // Init the shared memory of the reuslt grid to all zeros
    sharedMemoryInit(&g_resultData, g_dataLength);
}

static inline void gol_initMaster(unsigned int pattern, size_t worldWidth, size_t worldHeight)
{
    switch (pattern)
    {
    case 0:
        gol_initAllZeros(worldWidth, worldHeight);
        break;

    case 1:
        gol_initAllOnes(worldWidth, worldHeight);
        break;

    case 2:
        gol_initOnesInMiddle(worldWidth, worldHeight);
        break;

    case 3:
        gol_initOnesAtCorners(worldWidth, worldHeight);
        break;

    case 4:
        gol_initSpinnerAtCorner(worldWidth, worldHeight);
        break;

    default:
        printf("Pattern %u has not been implemented \n", pattern);
        exit(-1);
    }
}

// Swap the pointers of pA and pB.
static inline void gol_swap(unsigned char **pA, unsigned char **pB)
{
    unsigned char *temp = *pA;
    *pA = *pB;
    *pB = temp;
}

// Return the number of alive cell neighbors for data[x1+y1]
__device__ static inline unsigned int gol_countAliveCells(unsigned char *data,
                                                          size_t x0,
                                                          size_t x1,
                                                          size_t x2,
                                                          size_t y0,
                                                          size_t y1,
                                                          size_t y2)
{

    // Compute the number of alive cells by summing the states of each surrounding cell
    unsigned int aliveCellsCount = data[x0 + y0] +
                                   data[x1 + y0] +
                                   data[x2 + y0] +
                                   data[x0 + y1] +
                                   data[x2 + y1] +
                                   data[x0 + y2] +
                                   data[x1 + y2] +
                                   data[x2 + y2];

    return aliveCellsCount;
}

// Don't modify this function or your submitty autograding may incorrectly grade otherwise correct solutions.
static inline void gol_printWorld()
{
    int i, j;

    for (i = 0; i < g_worldHeight; i++)
    {
        printf("Row %2d: ", i);
        for (j = 0; j < g_worldWidth; j++)
        {
            printf("%u ", (unsigned int)g_data[(i * g_worldWidth) + j]);
        }
        printf("\n");
    }

    printf("\n\n");
}

// Main CUDA kernel function - handles parallel threading
__global__ void gol_kernel(unsigned char* d_data,
                           unsigned char* d_resultData,
                           unsigned int worldWidth,
                           unsigned int worldHeight)
{

    // Iterate over each cell of the grid
    for (unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < worldWidth * worldHeight;
         index += blockDim.x * gridDim.x) {

        // Use the thread index to calculate current x and y position
        size_t y = index / worldHeight;
        size_t x = index % worldWidth;

        // Compute offsets and perform analyses on cells
        // Set y0, y1 and y2
        size_t y0 = ((y + worldHeight - 1) % worldHeight) * worldWidth;
        size_t y1 = y * worldWidth;
        size_t y2 = ((y + 1) % worldHeight) * worldWidth;

        // Set x0, x1 and x2
        size_t x1 = x;
        size_t x0 = (x1 + worldWidth - 1) % worldWidth;
        size_t x2 = (x1 + 1) % worldWidth;

        // Call countAliveCells
        unsigned int aliveCellsCount = gol_countAliveCells(d_data, x0, x1, x2, y0, y1, y2);

        // Cell is currently alive
        if (d_data[x1 + y1]) {

            // Under-population (curr cell dies)
            if (aliveCellsCount < 2) {
                d_resultData[x1 + y1] = 0;
            }

            // Optimal population (curr cell survives)
            if (aliveCellsCount == 2 || aliveCellsCount == 3) {
                d_resultData[x1 + y1] = 1;
            }

            // Over-population (curr cell dies)
            if (aliveCellsCount > 3) {
                d_resultData[x1 + y1] = 0;
            }
        }

        // Cell is currently dead
        else {
            
            // Reproduction (curr cell becomes alive)
            if (aliveCellsCount == 3) {
                d_resultData[x1 + y1] = 1;
            }

            // Cell stays dead
            else {
                d_resultData[x1 + y1] = 0;
            }
        }
    }
}

// Launches the parallel computation of the world for a defined number of iterations
void gol_kernelLaunch(unsigned char** d_data,
                      unsigned char** d_resultData,
                      size_t worldWidth,
                      size_t worldHeight,
                      size_t iterationsCount,
                      ushort threadsCount)
{

    // Run the kernel for input num iterations over input num threads
    for (size_t i = 0; i < iterationsCount; i++) {

        // Compute block number based on CUDA docs (devblogs.nvidia.com -> see References "line 3")
        int blocksCount = ((g_worldWidth * g_worldHeight) + threadsCount - 1) / threadsCount;

        // Kernel call
        gol_kernel<<<blocksCount, threadsCount>>>(*d_data, *d_resultData, worldWidth, worldHeight);

        // Swap resultData and data arrays
        gol_swap(d_data, d_resultData);
    }

    // Need to call device synchronize before returning to main
    hipDeviceSynchronize();
} 

int main(int argc, char *argv[])
{
    unsigned int pattern = 0;
    unsigned int worldSize = 0;
    unsigned int iterations = 0;
    unsigned int threads = 0;
    unsigned int outputOn = 0;

    //printf("This is the Game of Life running in parallel on GPU(s).\n");

    if (argc != 6)
    {
        fprintf(stderr, "GOL requires 5 arguments: pattern number, sq size of the world, the number of iterations, the number of threads per block and if output is on e.g. ./gol 0 32 2 2 1\n");
        exit(-1);
    }

    pattern = atoi(argv[1]);
    worldSize = atoi(argv[2]);
    iterations = atoi(argv[3]);
    threads = atoi(argv[4]);
    outputOn = atoi(argv[5]);

    gol_initMaster(pattern, worldSize, worldSize);

    // Launches the parallel computation of the world for a defined number of iterations
    gol_kernelLaunch(&g_data, &g_resultData, g_worldWidth, g_worldHeight, iterations, threads);
                
    // Print statements for case of output on
    if (outputOn) {
        printf("######################### FINAL WORLD IS ###############################\n");                                                                  
        gol_printWorld();
    }

    // Return any memory to system
    hipFree(g_data);
    hipFree(g_resultData);

    return EXIT_SUCCESS;
}
